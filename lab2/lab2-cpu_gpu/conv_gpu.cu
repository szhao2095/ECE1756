#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <assert.h>
#include <iostream>
#include <iomanip>
#include <fstream>
#include <sstream>
#include <sys/time.h>
#include <unistd.h>
#include <cstdlib>

#include "utils.h"

// A variable that holds the number of CPU clock cycles per second
uint64_t clocks_per_sec;
// Declare the filter coefficient as read-only values that can be stored in the GPU constant memory
// for faster access than global memory.
__device__ __constant__ int device_filter[FILTER_SIZE*FILTER_SIZE*MAX_NUM_FILTERS];

/*********************************************************************************************
 * This function reads the time stamp using inline assembly code to measure code runtime.    *
 *********************************************************************************************/
inline uint64_t rdtsc(){
	uint32_t lo, hi;
	__asm__ volatile (".byte 0x0f, 0x31" : "=a" (lo), "=d" (hi));
	return (uint64_t)(((uint64_t)hi)<<32LL) | (uint64_t) lo;
}

/*********************************************************************************************
 * This is the CUDA 2D convolution kernel. It takes as an input pointers to the input and    *
 * output images in the GPU memory, along with the image dimensions. It divides up the image *
 * into tiles of size TILE_LOAD_SIZE x TILE_LOAD_SIZE, and loads each tile into the shared   *
 * memory where a group of threads can work on it.   		     			     *
 *********************************************************************************************/
__global__ void gpu_conv2d(int* input_image, int* output_image, int image_width, int image_height) {

	// Allocate space in shared memory to load an image tile
	__shared__ int image_tile[TILE_LOAD_SIZE][TILE_LOAD_SIZE];

	// Declare variables to use throughout the kernel
	int thread_id, tile_location_x, tile_location_y;
	int image_location_x, image_location_y, pixel_id;
	
	// Load tiles from Global memory to Shared memory for faster access
	for (int itr = 0; itr <= (TILE_LOAD_SIZE * TILE_LOAD_SIZE) / (TILE_SIZE * TILE_SIZE); itr++){
		// Calculate destination x and y indecies
		thread_id = (threadIdx.y * TILE_SIZE) + threadIdx.x + (itr * TILE_SIZE * TILE_SIZE);
		tile_location_y = thread_id / TILE_LOAD_SIZE;
		tile_location_x = thread_id % TILE_LOAD_SIZE;
		
		// Calculate source pixel index
		image_location_y = blockIdx.y * TILE_SIZE + tile_location_y - FILTER_RADIUS;
		image_location_x = blockIdx.x * TILE_SIZE + tile_location_x - FILTER_RADIUS;
		pixel_id = (image_location_y * image_width) + image_location_x;
		
		// Load pixels
		if (tile_location_y < TILE_LOAD_SIZE && tile_location_x < TILE_LOAD_SIZE){
			if (image_location_y >= 0 && image_location_y < image_height 
			    && image_location_x >= 0 && image_location_x < image_width){
				image_tile[tile_location_y][tile_location_x] = input_image[pixel_id];
			} else {
				image_tile[tile_location_y][tile_location_x] = 0;
			}
		}
	}
	__syncthreads();

	// Perform the 2D convolution
	int accum = 0;
	int y, x, z;
	z = blockIdx.z;
	for (y = 0; y < FILTER_SIZE; y++) {
		for (x = 0; x < FILTER_SIZE; x++) {
			accum += image_tile[threadIdx.y + y][threadIdx.x + x] * device_filter[(z * FILTER_SIZE * FILTER_SIZE) + (y * FILTER_SIZE) + x];
		}
	}

	// Write the output
	y = blockIdx.y * TILE_SIZE + threadIdx.y;
	x = blockIdx.x * TILE_SIZE + threadIdx.x;
	if (y < image_height && x < image_width){
		output_image[(z * image_width * image_height) + (y * image_width) + x] = accum;
	}
	__syncthreads();
}


int main(int argc, char *argv[]) {
	// Parse user command line arguments
	int num_filters;		// Number of filters for convolution
	std::string input_image_name;	// Name of input image
	if (argc < 3) {
		std::cout << "Missing command line arguments. Try running:" << std::endl;
		std::cout << "./conv_gpu <image_name> <num_filters>" << std::endl;
		return 0;
	}
	input_image_name = argv[1];
	num_filters = atoi(argv[2]);

	// Get the number of CPU cycles per second
	uint64_t start_clk = rdtsc();
	usleep(100000);
	uint64_t end_clk = rdtsc();
	clocks_per_sec = (end_clk - start_clk) * 10;

	// Declare pointers for all needed buffers
	int image_width; 		// Input image width
	int image_height; 		// Input image height
	int* host_input_image;		// Input image in CPU memory
	int* host_filter;		// Convolution filter in CPU memory
	int* host_output_image_gpu;	// GPU output image in CPU memory
	int* device_input_image;	// Input image in GPU memory
	int* device_output_image;	// GPU output image in GPU memory

	// Read filter data from file
	host_filter = (int*) malloc(sizeof(int) * FILTER_SIZE * FILTER_SIZE * num_filters);
	read_filters<int>("filters", FILTER_SIZE, host_filter, num_filters);

	// Read input image size
	if(!read_pgm_image_size(input_image_name, image_width, image_height)){
		return 0;
	}

	// Read image in PGM format
	host_input_image = (int*) malloc(sizeof(int) * image_width * image_height);
	read_pgm_image<int>(input_image_name, image_width, image_height, host_input_image, 0);

	// Allocate memory for GPU and CPU output images
	host_output_image_gpu = (int*) malloc(sizeof(int) * image_width * image_height * num_filters);
	
	// Allocate GPU memory for the input and output images
	cudaCheck(hipMalloc((void **) &device_input_image, image_width * image_height * sizeof(int)));
	cudaCheck(hipMalloc((void **) &device_output_image, image_width * image_height * num_filters * sizeof(int)));
	
	// Create CUDA grids and blocks with size depending on the tiles of the image
	dim3 dimGrid(((image_width-1)/TILE_SIZE)+1, ((image_height-1)/TILE_SIZE)+1, num_filters);
	dim3 dimBlock(TILE_SIZE, TILE_SIZE, 1);

	// Run the convolution for multiple times and get the average runtime
	double gpu_runtime_in_ms = 0;
	for(int run = 0; run < NUM_RUNS; run++){
		// Copy input image and convolution filter from host memory to GPU memory
	   	cudaCheck(hipMemcpy(device_input_image, host_input_image, image_width * image_height * sizeof(int), hipMemcpyHostToDevice));
		cudaCheck(hipMemcpyToSymbol(HIP_SYMBOL(device_filter), host_filter, FILTER_SIZE * FILTER_SIZE * num_filters * sizeof(int), 0));
		
		// Call the CUDA kernel to perform the 2D convolution on the GPU. Record time stamp before and after to measure runtime
		start_clk = rdtsc();
		gpu_conv2d<<<dimGrid,dimBlock>>>(device_input_image, device_output_image, image_width, image_height);
		hipDeviceSynchronize();
		end_clk = rdtsc();

		// Copy back the output image (i.e. result of convolution) from GPU memory to host memory
		cudaCheck(hipMemcpy(host_output_image_gpu, device_output_image, image_width * image_height * num_filters * sizeof(int), hipMemcpyDeviceToHost));
		
		gpu_runtime_in_ms += 1000.0 * (end_clk-start_clk) / clocks_per_sec;
	}
	gpu_runtime_in_ms /= NUM_RUNS;

	// Write the output images in PGM format
	for(int k = 0; k < num_filters; k++){
		std::ostringstream oss;
		oss << std::setw(2) << std::setfill('0') << k;
		std::string filename = std::string("out_gpu/") + oss.str() + std::string(".pgm");
		write_pgm_image<int>(filename, image_width, image_height, host_output_image_gpu + (k * image_width * image_height), 0);
	}

	// Print average GPU runtime
	std::cout << "Runtime for " << num_filters << " filter(s) averaged accross " << NUM_RUNS << " runs = " << gpu_runtime_in_ms << " ms" << std::endl;
	
	std::cout << "------------------------" << std::endl;

	// Free GPU memory
	hipFree(device_input_image);
	hipFree(device_output_image);
	hipFree(device_filter);
	
	// Free CPU memory
	free(host_filter);
	free(host_input_image);
	free(host_output_image_gpu);
	
	return 0;
}
